#include "hip/hip_runtime.h"
#include "IsingModel.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>

// Kernel CUDA para calcular las energías
__global__ void calculateEnergiesKernel(int* spins, int N, int J, int* energiesNonPeriodic, int* energiesPeriodic, int totalConfigs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalConfigs) return;

    int row, col, right, down;
    int energyNonPeriodic = 0, energyPeriodic = 0;

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            row = i;
            col = j;
            right = (i + 1) % N;  // Vecino a la derecha con condición periódica
            down = (j + 1) % N;   // Vecino de abajo con condición periódica

            // Energía no periódica
            if (i + 1 < N) {
                energyNonPeriodic -= J * spins[idx * N * N + i * N + j] * spins[idx * N * N + (i + 1) * N + j];
            }
            if (j + 1 < N) {
                energyNonPeriodic -= J * spins[idx * N * N + i * N + j] * spins[idx * N * N + i * N + (j + 1)];
            }

            // Energía periódica
            energyPeriodic -= J * spins[idx * N * N + i * N + j] * (spins[idx * N * N + right * N + j] + spins[idx * N * N + i * N + down]);
        }
    }

    // Guardar resultados en arrays globales
    energiesNonPeriodic[idx] = energyNonPeriodic;
    energiesPeriodic[idx] = energyPeriodic;
}

// Constructor
IsingModel::IsingModel(int N) : N(N) {
    // Inicializa las configuraciones de spins usando EnumerateIsing
    spins = createSpinsGrid();
}

// Destructor
IsingModel::~IsingModel() {
    // Nada que liberar en este caso
}

// Método para enumerar todas las configuraciones posibles de spins
std::vector<std::vector<std::vector<int>>> IsingModel::EnumerateIsing() const {
    std::vector<std::vector<std::vector<int>>> configurations;
    long long totalConfigs = 1LL << (N * N);  // 2^(N*N) configuraciones

    for (long long config = 0; config < totalConfigs; ++config) {
        std::vector<int> spinsFlat(N * N);
        for (int i = 0; i < N * N; ++i) {
            spinsFlat[i] = ((config >> i) & 1) == 1 ? 1 : -1;  // 1 para spin up, -1 para spin down.
        }

        // Convertir el vector plano en una matriz de NxN.
        std::vector<std::vector<int>> spinsGrid(N, std::vector<int>(N));
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                spinsGrid[i][j] = spinsFlat[i * N + j];
            }
        }

        configurations.push_back(spinsGrid);
    }

    return configurations;
}

// Función que calcula las energías usando enumeración de Ising y CUDA
void IsingModel::calculateEnergiesWithEnumerateIsingCUDA(int J, std::vector<int>& energiesNonPeriodic, std::vector<int>& energiesPeriodic) {
    std::vector<std::vector<std::vector<int>>> configurations = EnumerateIsing();
    long long totalConfigs = configurations.size();

    // Alocar memoria para almacenar spins en formato plano
    int* spinsFlat = new int[totalConfigs * N * N];
    for (long long config = 0; config < totalConfigs; ++config) {
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                spinsFlat[config * N * N + i * N + j] = configurations[config][i][j];
            }
        }
    }

    // Alocar memoria en GPU
    int *d_spins, *d_energiesNonPeriodic, *d_energiesPeriodic;
    hipMalloc(&d_spins, totalConfigs * N * N * sizeof(int));
    hipMalloc(&d_energiesNonPeriodic, totalConfigs * sizeof(int));
    hipMalloc(&d_energiesPeriodic, totalConfigs * sizeof(int));

    // Copiar spins a la GPU
    hipMemcpy(d_spins, spinsFlat, totalConfigs * N * N * sizeof(int), hipMemcpyHostToDevice);

    // Configurar tamaño de bloques e hilos
    int blockSize = 256;
    int numBlocks = (totalConfigs + blockSize - 1) / blockSize;

    // Lanzar kernel para calcular energías
    calculateEnergiesKernel<<<numBlocks, blockSize>>>(d_spins, N, J, d_energiesNonPeriodic, d_energiesPeriodic, totalConfigs);

    // Copiar los resultados a la CPU
    hipMemcpy(energiesNonPeriodic.data(), d_energiesNonPeriodic, totalConfigs * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(energiesPeriodic.data(), d_energiesPeriodic, totalConfigs * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memoria
    hipFree(d_spins);
    hipFree(d_energiesNonPeriodic);
    hipFree(d_energiesPeriodic);
    delete[] spinsFlat;
}
